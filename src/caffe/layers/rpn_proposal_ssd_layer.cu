#include "hip/hip_runtime.h"
/******************************************************************************

* Copyright 2018 The Apollo Authors. All Rights Reserved.

*

* Licensed under the Apache License, Version 2.0 (the License);

* you may not use this file except in compliance with the License.

* You may obtain a copy of the License at

*

* http://www.apache.org/licenses/LICENSE-2.0

*

* Unless required by applicable law or agreed to in writing, software

* distributed under the License is distributed on an AS IS BASIS,

* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.

* See the License for the specific language governing permissions and

* limitations under the License.

*****************************************************************************/

#include "caffe/layers/rpn_proposal_ssd_layer.hpp"
#include "caffe/util/dtout.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{

template <typename Dtype>
void RPNProposalSSDLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
        const vector<Blob<Dtype>*>& top)//check
{
    Dtype input_height = this->im_height_, input_width = this->im_width_;
    Dtype min_size_w_cur = this->min_size_w_;
    Dtype min_size_h_cur = this->min_size_h_;
    vector<Dtype> im_width_scale = vector<Dtype>(1, this->read_width_scale_);
    vector<Dtype> im_height_scale = vector<Dtype>(1, this->read_height_scale_);
    vector<Dtype> cords_offset_x = vector<Dtype>(1, Dtype(0));
    vector<Dtype> cords_offset_y = vector<Dtype>(1, this->read_height_offset_);
    CHECK_EQ(bottom.back()->count(1), 6);
    const Dtype* img_info_data = bottom.back()->cpu_data();
    input_width = img_info_data[0];
    input_height = img_info_data[1];
    CHECK_GT(input_width, 0);
    CHECK_GT(input_height, 0);
    im_width_scale.clear();
    im_height_scale.clear();
    cords_offset_x.clear();
    cords_offset_y.clear();
    for (int n = 0; n < bottom.back()->num(); n++) {
        im_width_scale.push_back(img_info_data[n * 6 + 2]);
        im_height_scale.push_back(img_info_data[n * 6 + 3]);
        CHECK_GT(im_width_scale[n], 0);
        CHECK_GT(im_height_scale[n], 0);
        cords_offset_x.push_back(img_info_data[n * 6 + 4]);
        cords_offset_y.push_back(img_info_data[n * 6 + 5]);
    }

    Dtype bsz01 = this->bbox_size_add_one_ ? Dtype(1.0) : Dtype(0.0);

    Dtype min_size_mode_and_else_or = true;
    if (this->min_size_mode_ == DetectionOutputSSDParameter_MIN_SIZE_MODE_HEIGHT_OR_WIDTH) {
        min_size_mode_and_else_or = false;
    } else {
        CHECK(this->min_size_mode_ == DetectionOutputSSDParameter_MIN_SIZE_MODE_HEIGHT_AND_WIDTH);
    }

    const int num = bottom[0]->num();
    const int map_height = bottom[0]->height();
    const int map_width  = bottom[0]->width();
    const Dtype heat_map_a = this->heat_map_a_vec_[0];
    const Dtype heat_map_b = this->heat_map_b_vec_[0];
    CHECK_EQ(bottom[0]->channels(), num_anchors_ * 2);
    CHECK_EQ(bottom[1]->num(), num);
    CHECK_EQ(bottom[1]->channels(), num_anchors_ * 4);
    CHECK_EQ(bottom[1]->height(), map_height);
    CHECK_EQ(bottom[1]->width(), map_width);

    const Dtype* prob_gpu_data = bottom[0]->gpu_data();
    const Dtype* tgt_gpu_data = bottom[1]->gpu_data();

    int num_bboxes = num_anchors_ * map_height * map_width;
    dt_conf_ahw_.Reshape(num_bboxes, 1, 1, 1);
    dt_bbox_ahw_.Reshape(num_bboxes, 4, 1, 1);

    vector<BBox<Dtype> > proposal_all;
    vector<vector<vector<Dtype> > > proposal_batch_vec(top.size());
    for (int i = 0; i < num; ++i) {
        //Timer tm;
        //tm.Start();
        rpn_cmp_conf_bbox_gpu(num_anchors_,
                map_height, map_width,
                input_height, input_width,
                heat_map_a, heat_map_b,
                this->allow_border_, this->allow_border_ratio_,
                min_size_w_cur, min_size_h_cur,
                min_size_mode_and_else_or, this->threshold_objectness_,
                bsz01, this->do_bbox_norm_,
                this->bbox_means_[0], this->bbox_means_[1],
                this->bbox_means_[2], this->bbox_means_[3],
                this->bbox_stds_[0], this->bbox_stds_[1],
                this->bbox_stds_[2], this->bbox_stds_[3],
                this->refine_out_of_map_bbox_, anc_.gpu_data(), 
                prob_gpu_data + bottom[0]->offset(i, 0, 0, 0), 
                tgt_gpu_data + bottom[0]->offset(i, 0, 0, 0), 
                dt_conf_ahw_.mutable_gpu_data(),
                dt_bbox_ahw_.mutable_gpu_data());
        //LOG(INFO)<<"nms rpn_cmp_conf_bbox time: "<<tm.MilliSeconds();
        //tm.Start();

        //do nms by gpu
        const Dtype* conf_data = dt_conf_ahw_.cpu_data();
        const Dtype* bbox_gpu_data = dt_bbox_ahw_.gpu_data();
        std::vector<int> indices;
        apply_nms_gpu(bbox_gpu_data, conf_data, num_bboxes, 4,
                Dtype(0.0), this->nms_max_candidate_n_[0], 
                this->nms_top_n_[0], this->nms_overlap_ratio_[0], 
                bsz01, &indices, overlapped_, idx_sm_, stream_,
                NULL, 1, 0, this->nms_gpu_max_n_per_time_);
        //LOG(INFO)<<"nms apply_nms_gpu time: "<<tm.MilliSeconds();

        const Dtype* bbox_data = dt_bbox_ahw_.cpu_data();
        if (top.size() == 0) {
            for (int k = 0; k < indices.size(); k++) {
                BBox<Dtype> bbox;
                bbox.id = i;
                int idk = indices[k];
                int idkx4 = idk * 4;
                bbox.score = conf_data[idk];
                int imid_cur = im_width_scale.size() > 1 ? i : 0;
                CHECK_LT(imid_cur, im_width_scale.size());
                bbox.x1 = bbox_data[idkx4] / im_width_scale[imid_cur] 
                    + cords_offset_x[imid_cur];
                bbox.y1 = bbox_data[idkx4 + 1] / im_height_scale[imid_cur] 
                    + cords_offset_y[imid_cur]; 
                bbox.x2 = bbox_data[idkx4 + 2] / im_width_scale[imid_cur] 
                    + cords_offset_x[imid_cur]; 
                bbox.y2 = bbox_data[idkx4 + 3] / im_height_scale[imid_cur] 
                    + cords_offset_y[imid_cur]; 
                proposal_all.push_back(bbox);
            }
        } else if (top.size() == 1) {
            for (int k = 0; k < indices.size(); k++) {
                vector<Dtype> bbox(6, 0);
                bbox[0] = i;
                int idk = indices[k];
                int idkx4 = idk * 4;
                bbox[1] = conf_data[idk];
                bbox[2] = bbox_data[idkx4];
                bbox[3] = bbox_data[idkx4 + 1];
                bbox[4] = bbox_data[idkx4 + 2];
                bbox[5] = bbox_data[idkx4 + 3];
                proposal_batch_vec[0].push_back(bbox);
            }
        } else {
            for (int k = 0; k < indices.size(); k++) {
                vector<Dtype> bbox(6, 0);
                bbox[0] = i;
                int idk = indices[k];
                int idkx4 = idk * 4;
                bbox[1] = conf_data[idk];
                bbox[2] = bbox_data[idkx4];
                bbox[3] = bbox_data[idkx4 + 1];
                bbox[4] = bbox_data[idkx4 + 2];
                bbox[5] = bbox_data[idkx4 + 3];
                Dtype bw = bbox[4] - bbox[2] + bsz01; 
                Dtype bh = bbox[5] - bbox[3] + bsz01; 
                Dtype bwxh = bw * bh;
                for(int t = 0; t < top.size(); t++) {
                    if(bwxh > this->proposal_min_area_vec_[t] 
                            && bwxh < this->proposal_max_area_vec_[t]) {
                        proposal_batch_vec[t].push_back(bbox);
                    }
                }
            }
        }
    }

    for(int t = 0; t < top.size(); t++) {
        if(proposal_batch_vec[t].empty()) {
            // for special case when there is no box
            top[t]->Reshape(1, rois_dim_, 1, 1);
            Dtype* top_boxes_scores = top[t]->mutable_cpu_data();
            caffe_set(top[t]->count(), Dtype(0), top_boxes_scores); 
        } else {
            const int top_num = proposal_batch_vec[t].size();
            top[t]->Reshape(top_num, rois_dim_, 1, 1);
            Dtype* top_boxes_scores = top[t]->mutable_cpu_data();
            for (int k = 0; k < top_num; k++) {
                top_boxes_scores[k*rois_dim_] = proposal_batch_vec[t][k][0];
                top_boxes_scores[k*rois_dim_+1] = proposal_batch_vec[t][k][2];
                top_boxes_scores[k*rois_dim_+2] = proposal_batch_vec[t][k][3];
                top_boxes_scores[k*rois_dim_+3] = proposal_batch_vec[t][k][4];
                top_boxes_scores[k*rois_dim_+4] = proposal_batch_vec[t][k][5];
                if (this->rpn_proposal_output_score_) {
                    top_boxes_scores[k*rois_dim_+5] = proposal_batch_vec[t][k][1];
                }
            }
        }
    }

    if (top.size() == 0) {
        for (int class_id = 0; class_id < this->num_class_; ++class_id) {
            this->output_bboxes_[class_id] = proposal_all;
        }
    }
}

template <typename Dtype>
void RPNProposalSSDLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, 
        const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {}

INSTANTIATE_LAYER_GPU_FUNCS(RPNProposalSSDLayer);

}  // namespace caffe
